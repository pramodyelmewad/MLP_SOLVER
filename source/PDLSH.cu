#include "hip/hip_runtime.h"
#include"stdio.h"
#include"stdlib.h"
/* Vector that holds  threads' vertex pair and corresponding latency */
struct min_dst_data
{
	long dst, i, j;
};

/* A kernel function to initialize min_dst_data vector */
__global__ void fill_dst_arr(struct min_dst_data *dst_arr, long dst, long sol)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id < sol)
		dst_arr[id].dst = dst;
}

/* Euclidean distance calculation */
__device__ __host__ long distD(int i,int j,float *x,float*y)
{
	float dx=x[i]-x[j];
	float dy=y[i]-y[j]; 
	return(sqrtf( (dx*dx) + (dy*dy) ));
}

/* A minimum triple finding kernel */
__global__ void find_min(struct min_dst_data *dst_tid, long sol, long i, long j)
{
	long id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id % j == 0 && (id + i) < sol)
	{
		if(dst_tid[id].dst > dst_tid[id + i].dst)
		{
			dst_tid[id].dst = dst_tid[id+i].dst;
			dst_tid[id].i = dst_tid[id+i].i;
			dst_tid[id].j = dst_tid[id+i].j;
		}
	}
}
/* A kernel for swap move evaluation using built-in reduction */
__global__ void swap(int *rt, long n, float *posx, float *posy, unsigned long long *dst_tid, long cost, long sol)
{
	long id = threadIdx.x + blockIdx.x * blockDim.x;
	long i, j;
	long change;
	if(id < sol)
	{
		i = n - 2 - floorf(((int)__dsqrt_rn(8*(sol - id - 1) + 1) - 1) / 2);
		j = id - i * (n - 1) + (i * (i + 1) / 2) + 1;
		if(i)
		{
			if(i == j-1 && j < n-1)
			{
				change = ((n-i)*distD(rt[i-1], rt[j], posx, posy)
					+(n-i-2)*distD(rt[i], rt[j+1], posx, posy))
					-
					 ((n-i)*distD(rt[i-1], rt[i], posx, posy)
					+(n-i-2)*distD(rt[j], rt[j+1], posx, posy));
			}
			else if(i == j-1 && j == n-1)
			{
				change = (n-i)*distD(rt[i-1], rt[j], posx, posy)
					-
					 (n-i)*distD(rt[i-1], rt[i], posx, posy);
			}
			else
			{
				change = ((n-i)*distD(rt[i-1], rt[j], posx, posy)
					+(n-i-1)*distD(rt[j], rt[i+1], posx, posy)
					+(n-j)*distD(rt[j-1], rt[i], posx, posy)
					+(n-j-1)*distD(rt[i], rt[j+1], posx, posy))
					-
					 ((n-i)*distD(rt[i-1], rt[i], posx, posy)
					+(n-i-1)*distD(rt[i], rt[i+1], posx, posy)
					+(n-j)*distD(rt[j-1], rt[j], posx, posy)
					+(n-j-1)*distD(rt[j], rt[j+1], posx, posy));
			}
			if(change < 0)
			{
				cost += change;
				atomicMin(dst_tid, ((unsigned long long)cost << 32) | id);
			}
		}
	}
}

/* A kernel for swap move evaluation using vector reduction */
__global__ void swap_loc(int *rt, long n, float *posx, float *posy, struct min_dst_data *dst_tid, long cost, long sol)
{
	long id = threadIdx.x + blockIdx.x * blockDim.x;
	long i, j;
	long change;
	__shared__ struct min_dst_data arr_dst[257];
	arr_dst[blockDim.x].dst = cost;
	if(threadIdx.x < blockDim.x)
		arr_dst[threadIdx.x].dst = cost;
	__syncthreads();
	if(id < sol)
	{

		i = n - 2 - floorf(((int)__dsqrt_rn(8*(sol - id - 1) + 1) - 1) / 2);
		j = id - i * (n - 1) + (i * (i + 1) / 2) + 1;
		if(i)
		{
			if(i == j-1 && j < n-1)
			{
				change = ((n-i)*distD(rt[i-1], rt[j], posx, posy)
					+(n-i-2)*distD(rt[i], rt[j+1], posx, posy))
					-
					 ((n-i)*distD(rt[i-1], rt[i], posx, posy)
					+(n-i-2)*distD(rt[j], rt[j+1], posx, posy));
			}
			else if(i == j-1 && j == n-1)
			{
				change = (n-i)*distD(rt[i-1], rt[j], posx, posy)
					-
					 (n-i)*distD(rt[i-1], rt[i], posx, posy);
			}
			else
			{
				change = ((n-i)*distD(rt[i-1], rt[j], posx, posy)
					+(n-i-1)*distD(rt[j], rt[i+1], posx, posy)
					+(n-j)*distD(rt[j-1], rt[i], posx, posy)
					+(n-j-1)*distD(rt[i], rt[j+1], posx, posy))
					-
					 ((n-i)*distD(rt[i-1], rt[i], posx, posy)
					+(n-i-1)*distD(rt[i], rt[i+1], posx, posy)
					+(n-j)*distD(rt[j-1], rt[j], posx, posy)
					+(n-j-1)*distD(rt[j], rt[j+1], posx, posy));
			}
			if(change < 0)
			{
				cost += change;
				arr_dst[threadIdx.x].dst = cost;
				arr_dst[threadIdx.x].i = i;
				arr_dst[threadIdx.x].j = j;
			}
		}
	}

	__syncthreads();
	int fact = blockDim.x % 2 == 0 ? blockDim.x >> 1 : (blockDim.x + 1) >> 1;
	while(fact)
	{
		if(threadIdx.x < fact)
		{
			if(arr_dst[threadIdx.x].dst > arr_dst[threadIdx.x + fact].dst)
			{
				arr_dst[threadIdx.x].dst = arr_dst[threadIdx.x + fact].dst;
				arr_dst[threadIdx.x].i = arr_dst[threadIdx.x + fact].i;
				arr_dst[threadIdx.x].j = arr_dst[threadIdx.x + fact].j;
			}
		}
		if(fact % 2 == 1 && fact != 1)
			fact++; 
		fact = fact / 2;
		__syncthreads();
	}
	__syncthreads();

	if(threadIdx.x == 0)
	{
		dst_tid[blockIdx.x].dst = arr_dst[0].dst;
		dst_tid[blockIdx.x].i = arr_dst[0].i;
		dst_tid[blockIdx.x].j = arr_dst[0].j;
	}
	
}
/* Device function used to calculate latency of solution after applying swap on i,j pair */
__device__ long get_route_dst(int*route, float *posx, float *posy, int i, int j, int n)
{
long ltcy = 0;
long d1 = 0, d2 = 0, d3 = 0;
	int x, y, z;
	for(x = 0, y =1; y <=i; x++, y++)
		d1 += (n - y) * distD(route[x], route[y], posx, posy);
	for(y = j + 1, x = i + 1; y < n; x = y, y++)
		d2 += (n - y) * distD(route[x], route[y], posx, posy);
	for( x = i, y = j, z = i; y > i; x = y, y--, z++)
	{
		d3 += (n - z -1) * distD(route[x], route[y], posx, posy);
	}
	ltcy = d1 + d2 + d3;
	return ltcy;
}
/* Function to arrange new solution using i,j pair */
void arrange_route(int*route, int i, int j, int n)
{
	int x, y;
	int * tmp;
	tmp = (int*)malloc(sizeof(int)*(j - i));
	for( x = 0, y = j; y > i; x++, y--)
		tmp[x] = route[y];
	for( x = i+1, y = 0; x <= j; x++, y++)
		route[x] = tmp[y];
	free(tmp);
}
/* Function to display the current solution */
__host__ __device__ void print_route(int *rt, int n)
{
	int i;
	printf("\nroute\n");
	for(i = 0; i < n; i++)
	printf("%d, ", rt[i]);
	printf("\n");
}
/* A kernel function for swap move evaluation using one-pass vector reduction */
__global__ void swap_loc_one(int *rt, long n, float *posx, float *posy, struct min_dst_data *dst_tid, long cost, long sol)
{
	long id = threadIdx.x + blockIdx.x * blockDim.x;
	long i, j;
	long change;
	if(id < sol)
	{
		i = n - 2 - floorf(((int)__dsqrt_rn(8*(sol - id - 1) + 1) - 1) / 2);
		j = id - i * (n - 1) + (i * (i + 1) / 2) + 1;
		if(i)
		{
			if(i == j-1 && j < n-1)
			{
				change = ((n-i)*distD(rt[i-1], rt[j], posx, posy)
					+(n-i-2)*distD(rt[i], rt[j+1], posx, posy))
					-
					 ((n-i)*distD(rt[i-1], rt[i], posx, posy)
					+(n-i-2)*distD(rt[j], rt[j+1], posx, posy));
			}
			else if(i == j-1 && j == n-1)
			{
				change = (n-i)*distD(rt[i-1], rt[j], posx, posy)
					-
					 (n-i)*distD(rt[i-1], rt[i], posx, posy);
			}
			else
			{
				change = ((n-i)*distD(rt[i-1], rt[j], posx, posy)
					+(n-i-1)*distD(rt[j], rt[i+1], posx, posy)
					+(n-j)*distD(rt[j-1], rt[i], posx, posy)
					+(n-j-1)*distD(rt[i], rt[j+1], posx, posy))
					-
					 ((n-i)*distD(rt[i-1], rt[i], posx, posy)
					+(n-i-1)*distD(rt[i], rt[i+1], posx, posy)
					+(n-j)*distD(rt[j-1], rt[j], posx, posy)
					+(n-j-1)*distD(rt[j], rt[j+1], posx, posy));
			}
			if(change < 0)
			{
				cost += change;
				dst_tid[id].dst = cost;
				dst_tid[id].i = i;
				dst_tid[id].j = j;
			}
		}
	}
}

/* A kernel function for two-opt move evaluation using one-pass vector reduction */
__global__ void two_opt_loc_one(int *rt, long n, float *posx, float *posy, struct min_dst_data *dst_tid, long cost, long sol)
{
	long i, j;
	long new_cost = cost;
	long id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id < sol)
	{
		i = n - 2 - floorf(((int)__dsqrt_rn(8*(sol - id - 1) + 1) - 1) / 2);
		j = id - i * (n - 1) + (i * (i + 1) / 2) + 1;
		if(i && i != j - 1)
		{
			new_cost = get_route_dst(rt, posx, posy, i, j, n);
			if(new_cost < cost)
			{
				dst_tid[id].dst = new_cost;
				dst_tid[id].i = i;
				dst_tid[id].j = j;
			}
			__syncthreads();
		}
	}
}
/* A kernel function for two-opt move evaluation using two-pass vector reduction */
__global__ void two_opt_loc(int *rt, long n, float *posx, float *posy, struct min_dst_data *dst_tid, long cost, long sol)
{
	long i, j;
	long new_cost = cost;
	long id = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ struct min_dst_data arr_dst[257];
	arr_dst[blockDim.x].dst = cost;
	if(threadIdx.x < blockDim.x)
		arr_dst[threadIdx.x].dst = cost;
	__syncthreads();
	if(id < sol)
	{

		i = n - 2 - floorf(((int)__dsqrt_rn(8*(sol - id - 1) + 1) - 1) / 2);
		j = id - i * (n - 1) + (i * (i + 1) / 2) + 1;
		if(i && i != j - 1)
		{
			new_cost = get_route_dst(rt, posx, posy, i, j, n);
			if(new_cost < cost)
			{
				arr_dst[threadIdx.x].dst = new_cost;
				arr_dst[threadIdx.x].i = i;
				arr_dst[threadIdx.x].j = j;
			}
		}
	}
	__syncthreads();
	int fact = blockDim.x % 2 == 0 ? blockDim.x >> 1 : (blockDim.x + 1) >> 1;
	while(fact)
	{
		if(threadIdx.x < fact)
		{
			if(arr_dst[threadIdx.x].dst > arr_dst[threadIdx.x + fact].dst)
			{
				arr_dst[threadIdx.x].dst = arr_dst[threadIdx.x + fact].dst;
				arr_dst[threadIdx.x].i = arr_dst[threadIdx.x + fact].i;
				arr_dst[threadIdx.x].j = arr_dst[threadIdx.x + fact].j;
			}
		}
		if(fact % 2 == 1 && fact != 1)
			fact++; 
		fact = fact / 2;
		__syncthreads();
	}
	__syncthreads();
	if(threadIdx.x == 0)
	{
		dst_tid[blockIdx.x].dst = arr_dst[0].dst;
		dst_tid[blockIdx.x].i = arr_dst[0].i;
		dst_tid[blockIdx.x].j = arr_dst[0].j;
	}
	__syncthreads();

}
/* A kernel function for two-opt move evaluation using one-pass vector reduction */
__global__ void two_opt(int *rt, long n, float *posx, float *posy, unsigned long long *dst_tid, long cost, long sol)
{
	long i, j;
	long new_cost;
	long id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id < sol)
	{
		i = n - 2 - floorf(((int)__dsqrt_rn(8*(sol - id - 1) + 1) - 1) / 2);
		j = id - i * (n - 1) + (i * (i + 1) / 2) + 1;
		if(i && i != j-1)
		{
			new_cost = get_route_dst(rt, posx, posy, i, j, n);
			if(new_cost < cost)
				atomicMin(dst_tid, ((unsigned long long)new_cost << 32) | id);
		}
	}
}

/* Initial solution construction based on NN */
long nn_route(int *route, long n, float *posx, float*posy)
{

	route[0] = 0;
	int k = 1, i = 0, j;
	float min;
	int minj, mini, count = 1, flag = 0;
	long ltcy = 0;
	int *visited = (int*)calloc(n,sizeof(int));
	visited[0] = 1;
	while(count!= n)
	{
		flag = 0;
		for(j = 1;j < n; j++)
		{
			if(i != j && !visited[j])
			{
				min = distD(i, j, posx,posy);
				minj = j;
				break;	
			}
		}

		for(j = minj+1; j < n; j++)
		{
			
			 if( !visited[j])
			{
				if(min > distD(i, j, posx, posy))
				{
					min = distD(i, j, posx, posy);
					mini = j;
					flag = 1;				
				}
			}
		}
		if(flag == 0)
			i = minj;
		else
			i = mini;
		route[k++] = i;
		visited[i] = 1;
		count++;
	}
	free(visited);
	for(i = 0, j = 1; j < n; i++, j++)
		ltcy += (n - j) * distD(route[i], route[j], posx, posy);
	return ltcy;
}
/* A function to verify the constructed solution is feasible or not */
void route_checker(int *route, int n)
{
	int i, *v, flag =0;
	v = (int*)calloc(n, sizeof(int));
	for(i = 0; i < n; i++)
		v[route[i]]++;
	for(i = 0; i < n; i++)
	{
		if(v[i] != 1)
		{
			printf("\nVisited counter: %d city Id: %d \n", v[i], i);
			flag = 1;
			break;
		}	
	}
	if(flag)
		printf("Invalid\t");
	else
		printf("Valid\t");
}
/* Single-thread reduction function to find the minimum triple values */
void find_min_cpu(struct min_dst_data *dst_tid, long sol)
{
	int min_i = 0, flag = 0;
	long minD = dst_tid[0].dst;
	for(int i = 1; i < sol; i++)
	{
		if(dst_tid[i].dst < minD)
		{
			minD = dst_tid[i].dst;
			min_i = i;
			flag = 1;
		}
	}
	if(flag)
	{
		dst_tid[0].dst = dst_tid[min_i].dst;
		dst_tid[0].i = dst_tid[min_i].i;
		dst_tid[0].j = dst_tid[min_i].j;
	}
}

int main(int argc, char *argv[])
{
	int ch, ch1, ch2, cnt, in1, n;
	float in2, in3;
	FILE *f;
	float *posx, *posy;
	float tm;
	char str[256];  
	long dst, ldst, loc_dst;
	int i, j, x, y, tmp, *route, flag, tid;
	int deviceId;
	clock_t start,end;
	
	hipGetDevice(&deviceId);
	f = fopen(argv[1], "r");
	if (f == NULL) {fprintf(stderr, "could not open file \n");  exit(-1);}
	char* p = strstr(argv[1], "TRP");

	start = clock();
	if(p)
	{
		fscanf(f, "%s\n", str);
		fscanf(f, "%s %d\n", str, &i);
		while(strcmp(str, "Number-of-machines:") != 0)
			fscanf(f, "%s %d\n", str, &i);
		n = i;n++;
		hipMallocManaged(&route, sizeof(int) * n);

		fscanf(f, "%s\n", str);
		while (strcmp(str, "y-Coor") != 0) 
			fscanf(f, "%s\n", str);

		cnt = 0;
		hipMallocManaged(&posx, sizeof(float) * n);
		if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
		hipMallocManaged(&posy, sizeof(float) * n);
		if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
		while (cnt < n) 
		{
			fscanf(f, "%d %f %f\n", &in1, &in2, &in3);
			posx[in1] = in2;
			posy[in1] = in3;
			cnt++;
		}
		fclose(f);
		printf("%s\t",argv[1]);
	}
	else
	{
		char buf[10];
		fscanf(f, "%s", buf);
		ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
		ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
		ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);

		ch = getc(f);  while ((ch != EOF) && (ch != ':')) ch = getc(f);
		fscanf(f, "%s\n", str);
		n = atoi(str);
		ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
		ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);


		cnt = 0;
		hipMallocManaged(&posx, sizeof(float) * n);
		if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
		hipMallocManaged(&posy, sizeof(float) * n);
		if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
		while (cnt < n) 
		{
			fscanf(f, "%d %f %f\n", &in1, &in2, &in3);
			posx[cnt] = in2;
			posy[cnt] = in3;
			cnt++;
		}
		fclose(f);
		printf("%s\t",argv[1]);
	}
	long sol = n * (n - 1) / 2;
	route = (int *)malloc(sizeof(int) * n);
	hipMallocManaged(&route, sizeof(int) * n);
	int blk, thrd;
	if(sol < 256)
	{
		blk = 1;
		thrd = sol;
	}
	else
	{
		blk = (sol - 1) / 256 + 1;
		thrd = 256;
	}

	dst = nn_route(route, n, posx, posy);
	printf("%ld\t",dst);
	route_checker(route, n);
	flag = 1;
	ldst = dst;
	struct min_dst_data * dst_arr;
	int fThrds, fBlks;
	printf("\nEnter reduction method\n1) Built-in Function\n2) Data Vector\n");
	scanf("%d", &ch1);
	switch(ch1)
	{
	case 1:
		unsigned long long *dst_tid;
		hipMallocManaged(&dst_tid, sizeof(unsigned long long));
		flag = 1;	
		ldst = dst;		
		while(flag)
		{
			flag = 0;
			*dst_tid = (((long)dst + 1) << 32) - 1;
			swap<<<blk, thrd>>>(route, n, posx, posy, dst_tid, dst, sol);
			hipDeviceSynchronize();
			loc_dst = *dst_tid >> 32;
			while(loc_dst < dst)
			{
				dst = loc_dst;
				tid = *dst_tid & ((1ull << 32) - 1); 
				x = n - 2 - floor((sqrt(8 * (sol - tid - 1) + 1) - 1) / 2);
				y = tid - x * (n - 1) + (x * (x + 1) / 2) + 1;
				tmp = route[x];
				route[x] = route[y];
				route[y] = tmp;
				*dst_tid = (((long)dst + 1) << 32) - 1;
				swap<<<blk, thrd>>>(route, n, posx, posy, dst_tid, dst, sol);
				hipDeviceSynchronize();
				loc_dst = *dst_tid >> 32;
			}
			*dst_tid = (((long)dst + 1) << 32) - 1;
			two_opt<<<blk, thrd>>>(route, n, posx, posy, dst_tid, dst, sol);
			hipDeviceSynchronize();
			loc_dst = *dst_tid >> 32;
			while(loc_dst < dst)
			{
				dst = loc_dst;
				tid = *dst_tid & ((1ull << 32) - 1); 
				x = n - 2 - floor((sqrt(8 * (sol - tid - 1) + 1) - 1) / 2);
				y = tid - x * (n - 1) + (x * (x + 1) / 2) + 1;
				*dst_tid = (((long)dst + 1) << 32) - 1;
				arrange_route(route, x, y, n);
				two_opt<<<blk, thrd>>>(route, n, posx, posy, dst_tid, dst, sol);
				hipDeviceSynchronize();
				loc_dst = *dst_tid >> 32;
			}
			if(dst < ldst)
			{
				flag = 1;
				ldst = dst;
			}
		}
		hipFree(dst_tid);
	break;

	case 2:
		printf("\nEnter reduction types\n1) Single Threaded Reduction\n2) One-pass Reduction\n3) Two-pass Reduction\n");
		scanf("%d", &ch2);
		switch(ch2)
		{
		case 1:
			hipMallocManaged(&dst_arr, sizeof(struct min_dst_data) * (blk + 1));
			dst_arr[blk].dst = dst;
			if (blk > 256)
			{
				fThrds = 256;
				fBlks = (blk - 1)/256 + 1;
			} 
			else
			{
				fThrds = blk;
				fBlks = 1;
			} 
			fill_dst_arr<<<fBlks, fThrds>>>(dst_arr, dst, blk);
			hipDeviceSynchronize();
			flag = 1;
			ldst = dst;
			while(flag)
			{
				flag = 0;
				swap_loc<<<blk, thrd>>>(route, n, posx, posy, dst_arr, dst, sol);
				hipDeviceSynchronize();
				find_min_cpu(dst_arr, blk);
				while(dst_arr[0].dst < dst)
				{
					dst = dst_arr[0].dst;
					x = dst_arr[0].i;
					y = dst_arr[0].j;
					tmp = route[x];
					route[x] = route[y];
					route[y] = tmp;
					swap_loc<<<blk, thrd>>>(route, n, posx, posy, dst_arr, dst, sol);
					hipDeviceSynchronize();
					find_min_cpu(dst_arr, blk);
				}
				two_opt_loc<<<blk, thrd>>>(route, n, posx, posy, dst_arr, dst, sol);
				hipDeviceSynchronize();
				find_min_cpu(dst_arr, blk);
				while(dst_arr[0].dst < dst)
				{
					dst = dst_arr[0].dst;
					x = dst_arr[0].i;
					y = dst_arr[0].j;
					arrange_route(route, x, y, n);
					two_opt_loc<<<blk, thrd>>>(route, n, posx, posy, dst_arr, dst, sol);
					hipDeviceSynchronize();
					find_min_cpu(dst_arr, blk);
				}
				if(dst < ldst)
				{
					ldst = dst;
					flag = 1;
				}
			}
		break;
		case 2:
			hipMallocManaged(&dst_arr, sizeof(struct min_dst_data) * (sol + 1));
			dst_arr[sol].dst = dst;
			fill_dst_arr<<<blk, thrd>>>(dst_arr, dst, sol);
			hipDeviceSynchronize();
			flag = 1;
			ldst = dst;
			while(flag)
			{
				flag = 0;
				dst_arr[sol].dst = dst;
				swap_loc_one<<<blk, thrd>>>(route, n, posx, posy, dst_arr, dst, sol);
				hipDeviceSynchronize();
				i = 1;
				j = 2;
				find_min<<<blk, thrd>>>(dst_arr, sol, i, j);
				hipDeviceSynchronize();
				i *= 2;
				j *= 2;
				while(i < sol)
				{
					find_min<<<blk, thrd>>>(dst_arr, sol, i, j);
					hipDeviceSynchronize();
					i *= 2;
					j *= 2;
				}
				while(dst_arr[0].dst < dst)
				{
					dst = dst_arr[0].dst;
					x = dst_arr[0].i;
					y = dst_arr[0].j;
					tmp = route[x];
					route[x] = route[y];
					route[y] = tmp;
					swap_loc_one<<<blk, thrd>>>(route, n, posx, posy, dst_arr, dst, sol);
					hipDeviceSynchronize();
					i = 1;
					j = 2;
					find_min<<<blk, thrd>>>(dst_arr, sol, i, j);
					hipDeviceSynchronize();
					i *= 2;
					j *= 2;
					while(i < sol)
					{
						find_min<<<blk, thrd>>>(dst_arr, sol, i, j);
						hipDeviceSynchronize();
						i *= 2;
						j *= 2;
					}
				}
				two_opt_loc_one<<<blk, thrd>>>(route, n, posx, posy, dst_arr, dst, sol);
				hipDeviceSynchronize();
				i = 1;
				j = 2;
				find_min<<<blk, thrd>>>(dst_arr, sol, i, j);
				hipDeviceSynchronize();
				i *= 2;
				j *= 2;
				while(i < sol)
				{
					find_min<<<blk, thrd>>>(dst_arr, sol, i, j);
					hipDeviceSynchronize();
					i *= 2;
					j *= 2;
				}
				while(dst_arr[0].dst < dst)
				{
					dst = dst_arr[0].dst;
					x = dst_arr[0].i;
					y = dst_arr[0].j;
					arrange_route(route, x, y, n);
					two_opt_loc_one<<<blk, thrd>>>(route, n, posx, posy, dst_arr, dst, sol);
					hipDeviceSynchronize();
					i = 1;
					j = 2;
					find_min<<<blk, thrd>>>(dst_arr, sol, i, j);
					hipDeviceSynchronize();
					i *= 2;
					j *= 2;
					while(i < sol)
					{
						find_min<<<blk, thrd>>>(dst_arr, sol, i, j);
						hipDeviceSynchronize();
						i *= 2;
						j *= 2;
					}
				}
				if(dst < ldst)
				{
					ldst = dst;
					flag = 1;
				}
			}
		break;
		case 3:
			hipMallocManaged(&dst_arr, sizeof(struct min_dst_data) * (blk + 1));
			dst_arr[blk].dst = dst;
			int fThrds, fBlks;
			if (blk > 256)
			{
				fThrds = 256;
				fBlks = (blk - 1)/256 + 1;
			} 
			else
			{
				fThrds = blk;
				fBlks = 1;
			} 
			fill_dst_arr<<<fBlks, fThrds>>>(dst_arr, dst, blk);
			hipDeviceSynchronize();
			while(flag)
			{
				flag = 0;
				swap_loc<<<blk, thrd>>>(route, n, posx, posy, dst_arr, dst, sol);
				hipDeviceSynchronize();
				i = 1;
				j = 2;
				find_min<<<fBlks, fThrds>>>(dst_arr, blk, i, j);
				hipDeviceSynchronize();
				i *= 2;
				j *= 2;
				while(i < blk)
				{
					find_min<<<fBlks, fThrds>>>(dst_arr, blk, i, j);
					hipDeviceSynchronize();
					i *= 2;
					j *= 2;
				}
				while(dst_arr[0].dst < dst)
				{
					dst = dst_arr[0].dst;
					x = dst_arr[0].i;
					y = dst_arr[0].j;
					tmp = route[x];
					route[x] = route[y];
					route[y] = tmp;
					swap_loc<<<blk, thrd>>>(route, n, posx, posy, dst_arr, dst, sol);
					hipDeviceSynchronize();
					i = 1;
					j = 2;
					find_min<<<fBlks, fThrds>>>(dst_arr, blk, i, j);
					hipDeviceSynchronize();
					i *= 2;
					j *= 2;
					while(i < blk)
					{
						find_min<<<fBlks, fThrds>>>(dst_arr, blk, i, j);
						hipDeviceSynchronize();
						i *= 2;
						j *= 2;
					}
				}
				two_opt_loc<<<blk, thrd>>>(route, n, posx, posy, dst_arr, dst, sol);
				hipDeviceSynchronize();
				i = 1;
				j = 2;
				find_min<<<fBlks, fThrds>>>(dst_arr, blk, i, j);
				hipDeviceSynchronize();
				i *= 2;
				j *= 2;
				while(i < blk)
				{
					find_min<<<fBlks, fThrds>>>(dst_arr, blk, i, j);
					hipDeviceSynchronize();
					i *= 2;
					j *= 2;
				}
				while(dst_arr[0].dst < dst)
				{
					dst = dst_arr[0].dst;
					x = dst_arr[0].i;
					y = dst_arr[0].j;
					arrange_route(route, x, y, n);
					hipDeviceSynchronize();
					two_opt_loc<<<blk, thrd>>>(route, n, posx, posy, dst_arr, dst, sol);
					hipDeviceSynchronize();
					i = 1;
					j = 2;
					find_min<<<fBlks, fThrds>>>(dst_arr, blk, i, j);
					hipDeviceSynchronize();
					i *= 2;
					j *= 2;
					while(i < blk)
					{
						find_min<<<fBlks, fThrds>>>(dst_arr, blk, i, j);
						hipDeviceSynchronize();
						i *= 2;
						j *= 2;
					}
				}
				if(dst < ldst)
				{
					ldst = dst;
					flag = 1;
				}
			}
		break;
		}
	break;
	}
	end = clock();
	tm = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("%ld\t%f\n",dst,tm);
	hipFree(posx);
	hipFree(dst_arr);
	hipFree(posy);
	hipFree(route);
	return 0;
}
